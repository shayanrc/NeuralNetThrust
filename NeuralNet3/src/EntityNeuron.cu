#include "hip/hip_runtime.h"
/*
 * EntityNeuron.cpp
 *
 *  Created on: 19-Mar-2014
 *      Author: xion
 */

#include "EntityNeuron.h"

EntityNeuron::EntityNeuron(std::string entName,int no_of_inputs):Neuron(no_of_inputs)
{
	std::cout<<"Making EntityNeuron ["<<entName<<"] with "<<no_of_inputs<<" inputs\n";
	entityName=entName;
	inputs=std::vector<double>(no_of_inputs,0);
	learnigRate=1;
	momentumConstant=0;

}

template< typename T >
struct multiplyByConstant
{
    const T constant;

    multiplyByConstant(T _constant) : constant(_constant) {}

    __host__ __device__
        void operator()( T& VecElem) const {
            VecElem=VecElem*constant;
        }
};


double activiation(double x)
{
	return 1/(1+exp(-x));
}

double deactiviation(double x)
{
	return x*(1-x);
}

void EntityNeuron::setInputs(std::vector<double> entityInputs)
{
	inputs=entityInputs;

	//added for bias
	inputs.push_back(1);
}

double EntityNeuron::feedForward()
{
	std::cout<<"EntityNeuron::feedForward : Inputs ("<<entityName<<") :\n";
	printVector(inputs);
	//create device vector and store inputs in it
	thrust::device_vector<double> d_inputs(inputs.size());
	thrust::copy(inputs.begin(),inputs.end(),d_inputs.begin());
	thrust::device_vector<double> WxI(inputCount);
	thrust::transform(d_inputs.begin(),d_inputs.end(),d_weightVector.begin(),WxI.begin(),thrust::multiplies<double>());

	double output=thrust::reduce(WxI.begin(),WxI.end());
	std::cout<<"Output["<<entityName<<"] :"<<output<<"\n";
	return output;
}

void EntityNeuron::setLearnigRate(double rate)
{
	learnigRate=rate;
}

void EntityNeuron::modifyweights(double dw)
{
	thrust::device_vector<double> weightChange(inputs.size());

	//Calculate weightChange
	//weightChange=inputs*dw*learningRate
	thrust::copy(inputs.begin(),inputs.end(),weightChange.begin());
	thrust::for_each(weightChange.begin(),weightChange.end(),multiplyByConstant<double>(learnigRate));
	thrust::for_each(weightChange.begin(),weightChange.end(),multiplyByConstant<double>(dw));

	thrust::transform(weightChange.begin(),weightChange.end(),d_momentum.begin(),weightChange.begin(),thrust::plus<double>());
	thrust::transform(d_weightVector.begin(),d_weightVector.end(),weightChange.begin(),d_weightVector.begin(),thrust::plus<double>());

	//store the change inweight as the next iterations momentum
	d_momentum=weightChange;
	thrust::for_each(d_momentum.begin(),d_momentum.end(),multiplyByConstant<double>(momentumConstant));

}

std::string EntityNeuron::getName()
{
	return entityName;
}

EntityNeuron::~EntityNeuron()
{
	// TODO Auto-generated destructor stub
}

