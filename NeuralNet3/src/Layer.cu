#include "hip/hip_runtime.h"
/*
 * Layer.cpp
 *
 *  Created on: 04-Dec-2013
 *      Author: xion
 */

#include "Layer.h"

template< typename T >
struct multiplyByConstant
{
    const T constant;

    multiplyByConstant(T _constant) : constant(_constant) {}

    __host__ __device__
        double operator()( T& VecElem) const {
            return VecElem*constant;
        }
};


Layer::Layer(int input_size, int output_size)
{
	init(input_size,output_size);

	learningRate=1;
	momentumConstant=0;
}


Layer::Layer(int input_size, int output_size,double _learningRate,double _momentumConstant)
{
	init(input_size,output_size);
	learningRate=_learningRate;
	momentumConstant=_momentumConstant;
}

void Layer::init(int input_size, int output_size)
{
	no_of_inputs=input_size+1;
	no_of_neurons=output_size;

	neurons=std::vector<Neuron>(no_of_neurons,Neuron(no_of_inputs));

	h_inputs=thrust::host_vector<double>(no_of_inputs);
	h_outputs=thrust::host_vector<double>(no_of_inputs,0);
	h_desired_outputs=thrust::host_vector<double>(no_of_neurons);
	h_dW=thrust::host_vector<double>(no_of_neurons);

	d_inputs=thrust::device_vector<double>(no_of_inputs);
	d_outputs=thrust::device_vector<double>(no_of_neurons);
	d_desired_outputs=thrust::device_vector<double>(no_of_neurons);
	d_dW=thrust::device_vector<double>(no_of_neurons);
	randomizeWeights();
}

void Layer::randomizeWeights()
{
	for(int neuronIndex=0; neuronIndex<neurons.size();neuronIndex++)
	{

		//std::cout<<"neuron :"<<neuronIndex<<" Randomize Weights\n";
		neurons[neuronIndex].randomizeWeights();
	}
}

//functor for sigmoid activation function
template< typename T >
struct activation
	{
		__host__ __device__ void operator()( T &x ) const {
		x = 1/(1+exp(-x));
	}
};

//functor for calculating derivative of sigmoid activation function
//template< typename T >
struct deactivation
	{
		__host__ __device__ void operator()( double &x ) const {
		x = x*(1-x);
	}
};

//Activation(Sum(Weight x Input))
void Layer::feedForward()
{

	//std::cout<<"Inside feedForward";
	thrust::device_vector<double> d_WxI=thrust::device_vector<double>(d_inputs.size());

	//std::cout<<"\nInputs : "<<d_inputs.size();
	//printVector(d_inputs);

	for(int neuronIndex=0;neuronIndex<no_of_neurons;neuronIndex++)
	{
		//d_inputs=h_inputs;

		//d_WxI=thrust::device_vector<double>(neurons[neuronIndex].d_weightVector.size());

		//std::cout<<"\nNeuron ["<<neuronIndex<<"] : feedforward :: "<<neurons[neuronIndex].d_weightVector.size()<<"\n";
		//printVector(neurons[neuronIndex].d_weightVector);




		//d_WxI = d_inputs * neurons[neuronIndex].d_weightVector;
		thrust::transform(d_inputs.begin(),d_inputs.end(),neurons[neuronIndex].d_weightVector.begin(),d_WxI.begin(),thrust::multiplies<double>());

//		std::cout<<"d_WxI :";
//		printVector(d_WxI);


		//d_outputs[neuronIndex] = sum(d_WxI)
		d_outputs[neuronIndex]=thrust::reduce(d_WxI.begin(),d_WxI.end());

		//std::cout<<"\nNeuron ["<<neuronIndex<<"] Output: "<<d_outputs[neuronIndex];


	}

	std::cout<<"\nfeedForward outputs before activation : ";
	printVector(d_outputs);
	//d_outputs=activation(d_outputs)
	thrust::for_each( d_outputs.begin() , d_outputs.end() , activation<double>() );
	std::cout<<"\nfeedForward outputs after activation : ";
	printVector(d_outputs);
}



void Layer::calculateError()
{
	std::cout<<"In calculate error\n";
	thrust::transform(d_desired_outputs.begin(),d_desired_outputs.end(),d_outputs.begin(),d_dW.begin(),thrust::minus<double>());

}


//Theres a bug here: if no of inputs and no of outputs are not same this fails TODO Fixit
void Layer::propagateBack(Layer *layer)
{
	thrust::device_vector<double> d_WxdW(no_of_inputs,0);
	thrust::fill(layer->d_dW.begin(),layer->d_dW.end(),0);
	for(unsigned int neuronIndex=0;neuronIndex<layer->d_dW.size();neuronIndex++)
	{
		//d_WxdW = d_dW * d_WxdW
		//thrust::transform(d_dW.begin(),d_dW.end(),neurons[neuronIndex].d_weightVector.begin(),d_WxdW.begin(),thrust::multiplies<double>());
		thrust::transform(neurons[neuronIndex].d_weightVector.begin(),neurons[neuronIndex].d_weightVector.end(),d_WxdW.begin(),multiplyByConstant<double>(d_dW[neuronIndex]));

		thrust::transform(d_WxdW.begin(),d_WxdW.end(),layer->d_dW.begin(),layer->d_dW.begin(),thrust::plus<double>());

		//prev_dW[i] = sum(d_WxdW)
		//layer.d_dW[neuronIndex]=thrust::reduce(d_WxdW.begin(),d_WxdW.end());
	}


	std::cout<<"\nd_dW in propagateBack: ";
	printVector(d_dW);
}

//Function to set the outputs as inputs of the next layer
//Recheck this function
void Layer::propagateOutput(Layer *layer)
{
	layer->d_inputs=d_outputs;

	//Add another input with value 1 for bias
	layer->d_inputs.push_back(1);

	std::cout<<"\npropagated Output: ";
	printVector(layer->d_inputs);
}

template< typename T >
struct modifyweight_functor
{
    const T learningRate;
    const T dW;

    modifyweight_functor(T _learningRate,T _dW) : learningRate(_learningRate),dW(_dW) {}

    __host__ __device__
        float operator()(const T& W, const T& I) const {
            return W+learningRate*dW*I;
        }
};

template< typename T >
struct calculateWeightChange
{
    const T learningRate;
    const T dW;

    calculateWeightChange(T _learningRate,T _dW) : learningRate(_learningRate),dW(_dW) {}

    __host__ __device__
        float operator()(const T& I) const {
            return learningRate*dW*I;
        }
};


template< typename T >
struct modifymomentum_functor
{
    //const T learningRate;
    const T momentumConstant;

    modifymomentum_functor(T _momentumConstant) : momentumConstant(_momentumConstant) {}

    __host__ __device__
        float operator()(const T& weightChange) const {
            return momentumConstant*weightChange;
        }
};

void Layer::modifyweights()
{

	thrust::device_vector<double> d_WeightChange=thrust::device_vector<double>(no_of_inputs);

	thrust::device_vector<double> d_deActivatedOutput=thrust::device_vector<double>(d_outputs);

	//thrust::device_vector<double> d_learningRate=thrust::device_vector<double>(no_of_inputs,learningRate);

	//Apply inverse activation function to the output
	thrust::for_each(d_deActivatedOutput.begin(),d_deActivatedOutput.end(),deactivation());

	std::cout<<"d_deActivatedOutput : ";
	printVector(d_deActivatedOutput);

	std::cout<<"d_dW : ";
	printVector(d_dW);

	//dW[neuronIndex]=deActivation(output[neuronIndex])*dW[neuronIndex];
	thrust::transform(d_deActivatedOutput.begin(),d_deActivatedOutput.end(),d_dW.begin(),d_dW.begin(),thrust::multiplies<double>());

	//std::cout<<"d_WeightChange : ";
	//printVector(d_WeightChange);

	//For each neuron:
	for(int neuronIndex=0;neuronIndex<no_of_neurons;neuronIndex++)
	{
		//TODO recheck the momentum logic I dont think it;s being done right
		//weightChange[neuronIndex][inputIndex]= LEARNING_RATE * dW[neuronIndex] * input[inputIndex];
		//weight[neuronIndex][inputIndex] = weight[neuronIndex][inputIndex]+weightChange[neuronIndex][inputIndex]

		/*std::cout<<"Neuron "<<neuronIndex<<" Weights before modifyweight: ";
		printVector(neurons[neuronIndex].d_weightVector);*/
		thrust::transform(d_inputs.begin(),d_inputs.end(),d_WeightChange.begin(),calculateWeightChange<double>(learningRate,d_dW[neuronIndex]));
		thrust::transform(d_WeightChange.begin(),d_WeightChange.end(),neurons[neuronIndex].d_weightVector.begin(),neurons[neuronIndex].d_weightVector.begin(),thrust::plus<double>());

		/*std::cout<<"Neuron "<<neuronIndex<<" Weights after modifyweight: ";
		printVector(neurons[neuronIndex].d_weightVector);*/

		//weight[neuronIndex][inputIndex] = weight[neuronIndex][inputIndex]+momentum[neuronIndex][inputIndex];
		thrust::transform(neurons[neuronIndex].d_momentum.begin(),neurons[neuronIndex].d_momentum.end(),neurons[neuronIndex].d_weightVector.begin(),neurons[neuronIndex].d_weightVector.begin(),thrust::plus<double>());


		//momentum[weightIndex]=MOMENTUM_CONSTANT*weightChange[weightIndex];
		thrust::transform(d_WeightChange.begin(),d_WeightChange.end(),neurons[neuronIndex].d_momentum.begin(),modifymomentum_functor<double>(momentumConstant));

	}

}

double Layer::getLearningRate() const {
	return learningRate;
}

void Layer::setLearningRate(double learningRate) {
	this->learningRate = learningRate;
}

double Layer::getMomentumConstant() const {
	return momentumConstant;
}

void Layer::setMomentumConstant(double momentumConstant) {
	this->momentumConstant = momentumConstant;
}

int Layer::getNoOfInputs() const {
	return no_of_inputs;
}

int Layer::getNoOfNeurons() const {
	return no_of_neurons;
}

/*thrust:: host_vector<double> Layer::getDesiredOutputs()
{
	return	h_desired_outputs;
}*/


std::vector<double> Layer::getDesiredOutputs(){

	std::vector<double> outputs(no_of_neurons);

	thrust::copy(h_desired_outputs.begin(),h_desired_outputs.end(),outputs.begin());
	return outputs;
}

std::vector<double> Layer::getinputs()
{
	std::vector<double> inputs(no_of_inputs);
	thrust::copy(h_inputs.begin(),h_inputs.end(),inputs.begin());
	return inputs;
}


/*thrust:: host_vector<double> Layer::getinputs()
{
	return h_inputs;
}*/


std::vector<double> Layer::getError()
{
	std::vector<double> error(no_of_neurons);
	h_dW=d_dW;
	thrust::copy(h_dW.begin(),h_dW.end(),error.begin());
	return error;
}

/*thrust:: host_vector<double> Layer::getError()
{
	h_dW=d_dW;
	return h_dW;
}*/

void Layer::setInputs(std::vector<double> inputs){

	h_inputs=inputs;

	//Add 1 to the end for bias
	h_inputs.push_back(1);

	//Copy to device
	d_inputs=h_inputs;

//	std::cout<<"Set inputs: ";
//	printVector(d_inputs);

}

void Layer::setInputs(thrust::host_vector<double> inputs){

	h_inputs=inputs;

	//Add 1 to the end for bias
	h_inputs.push_back(1);

	//Copy to device
	d_inputs=h_inputs;
}

void Layer::setDesiredOutputs(std::vector<double> outputs){


	h_desired_outputs=outputs;

	//Copy to device
	d_desired_outputs=h_desired_outputs;
}

/*thrust::host_vector<double> Layer::getOutputs(){

	h_outputs=d_outputs;
	return h_outputs;
}*/

std::vector<double> Layer::getOutputs(){
	//std::cout<<"Inside getOutputs";
	std::vector<double> outputs(no_of_neurons);
	h_outputs=d_outputs;
	//std::cout<<"Copied from device to host";
	thrust::copy(d_outputs.begin(),d_outputs.end(),outputs.begin());
	//std::cout<<"Copied from device to std";
	return outputs;
}

void Layer::identify()
{
	std::cout<<"Generic Layer!!";
}
