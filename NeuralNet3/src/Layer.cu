#include "hip/hip_runtime.h"
/*
 * Layer.cpp
 *
 *  Created on: 04-Dec-2013
 *      Author: xion
 */

#include "Layer.h"

Layer::Layer(int input_size, int output_size)
{
	no_of_inputs=input_size;
	no_of_neurons=output_size;

	neurons=std::vector<Neuron>(no_of_neurons,Neuron(no_of_inputs));

	h_inputs=thrust::host_vector<double>(no_of_inputs);
	h_outputs=thrust::host_vector<double>(no_of_inputs,0);
	h_desired_outputs=thrust::host_vector<double>(no_of_neurons);
	h_dW=thrust::host_vector<double>(no_of_neurons);

	d_inputs=thrust::device_vector<double>(no_of_inputs);
	d_outputs=thrust::device_vector<double>(no_of_neurons);
	d_desired_outputs=thrust::device_vector<double>(no_of_neurons);
	d_dW=thrust::device_vector<double>(no_of_neurons);
}

//functor for sigmoid activation function
template< typename T >
struct activation
	{
		__host__ __device__ void operator()( T &x ) const {
		x = 1/(1+exp(-x));
	}
};

//functor for calculating derivative of sigmoid activation function
template< typename T >
struct deactivation
	{
		__host__ __device__ void operator()( T &x ) const {
		x = x*(1-x);
	}
};

//Activation(Sum(Weight x Input))
void Layer::feedForward()
{
	thrust::device_vector<double> d_WxI;

	for(int neuronIndex=0;neuronIndex<no_of_neurons;neuronIndex++)
	{
		d_inputs=h_inputs;

		//d_WxI = neurons[neuronIndex].d_weightVector;
		d_WxI=thrust::device_vector<double>(neurons[neuronIndex].d_weightVector.begin(),neurons[neuronIndex].d_weightVector.end());

		//d_WxI = d_inputs * dWxI
		thrust::transform(d_inputs.begin(),d_inputs.end(),d_WxI.begin(),d_WxI.end(),thrust::multiplies<double>());

		//d_outputs[neuronIndex] = sum(d_WxI)
		d_outputs[neuronIndex]=thrust::reduce(d_WxI.begin(),d_WxI.end());
	}

	//d_outputs=activation(d_outputs)
	thrust::for_each( d_outputs.begin() , d_outputs.end() , activation<double>() );

}



void Layer::calculateError()
{
	thrust::transform(d_desired_outputs.begin(),d_desired_outputs.end(),d_outputs.begin(),d_dW.begin(),thrust::minus<double>());

}

void Layer::propagateBack(thrust::device_vector<double> prev_dW)
{
	thrust::device_vector<double> d_WxdW;

	for(int neuronIndex=0;neuronIndex<prev_dW.size();neuronIndex++)
		{
			//d_WxdW = neurons[i].d_weightVector;

			d_WxdW=thrust::device_vector<double>(neurons[neuronIndex].d_weightVector.begin(),neurons[neuronIndex].d_weightVector.end());

			//d_WxdW = d_inputs * d_WxdW
			thrust::transform(d_dW.begin(),d_dW.end(),d_WxdW.begin(),d_WxdW.end(),thrust::multiplies<double>());

			//prev_dW[i] = sum(d_WxdW)
			prev_dW[neuronIndex]=thrust::reduce(d_WxdW.begin(),d_WxdW.end());
		}

}

template< typename T >
struct modifyweight_functor
{
    const T learningRate;
    const T dW;

    modifyweight_functor(T _learningRate,T _dW) : learningRate(_learningRate),dW(_dW) {}

    __host__ __device__
        float operator()(const T& W, const T& I) const {
            return W-learningRate*dW*I;
        }
};

template< typename T >
struct modifymomentum_functor
{
    //const T learningRate;
    const T momentumConstant;

    modifymomentum_functor(T _momentumConstant) : momentumConstant(_momentumConstant) {}

    __host__ __device__
        float operator()(const T& weightChange) const {
            return momentumConstant*weightChange;
        }
};

void Layer::modifyweights()
{

	thrust::device_vector<double> d_WeightChange=thrust::device_vector<double>(no_of_inputs);

	thrust::device_vector<double> d_deActivatedOutput=thrust::device_vector<double>(d_outputs);

	thrust::device_vector<double> d_learningRate=thrust::device_vector<double>(no_of_inputs,learningRate);

	thrust::for_each(d_deActivatedOutput.begin(),d_deActivatedOutput.end(),deactivation<double>());

	//dW[neuronIndex]=deActivation(output[neuronIndex])*dW[neuronIndex];
	thrust::transform(d_deActivatedOutput.begin(),d_deActivatedOutput.end(),d_dW.begin(),d_dW.end(),thrust::multiplies<double>());


	for(int neuronIndex=0;neuronIndex<no_of_neurons;neuronIndex++)
	{
		//weightChange[neuronIndex][inputIndex]= LEARNING_RATE * dW[neuronIndex] * input[inputIndex];
		//weight[neuronIndex][inputIndex] = weight[neuronIndex][inputIndex]+weightChange[neuronIndex][inputIndex]
		thrust::transform(d_WeightChange.begin(),d_WeightChange.end(),neurons[neuronIndex].d_weightVector.begin(),neurons[neuronIndex].d_weightVector.begin(),modifyweight_functor<double>(learningRate,d_dW[neuronIndex]));

		//weight[neuronIndex][inputIndex] = weight[neuronIndex][inputIndex]+momentum[neuronIndex][inputIndex];
		thrust::transform(neurons[neuronIndex].d_momentum.begin(),neurons[neuronIndex].d_momentum.end(),neurons[neuronIndex].d_weightVector.begin(),neurons[neuronIndex].d_weightVector.begin(),thrust::plus<double>());

		//momentum[weightIndex]=MOMENTUM_CONSTANT*weightChange[weightIndex];
		thrust::transform(d_inputs.begin(),d_inputs.end(),neurons[neuronIndex].d_momentum.begin(),modifymomentum_functor<double>(momentumConstant));
	}

}
